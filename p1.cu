
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <vector>
#include <string>

using namespace std;

void printGrid(bool* grid, int w, int h) {
    // cout << w << h;
    for (int i = 0; i < h; i++) {
        for (int j = 0; j < w; j++) {
            if (grid[i*w +j]) cout << "#";
            else cout << ".";
        }
        cout << endl;
    }
}


bool liveOrDie(bool* grid, int w, int h, int i, int j, int A, int B, int C){
    // get number of alive neighbours
    int sum = 0;
    int n, m;
    for (int k = -1; k <= 1; ++k) {
        for (int l = -1; l <= 1; ++l) {
            n = i + k;
            m = j + l;
            if (i==n && j==m) continue;
            n += h;
            m += w;
            sum += grid[(n%h)*w + m%w];
            // cout << i << " " << j;
            // printf("Here\n"); fflush(stdout);
        }
    }

    
    
    // decide whether cell should live or die
    bool live = grid[i*w + j];
    if (grid[i*w + j]) {
        if (sum > B || sum < A) live = false;
    } else {
        if (sum == C) live = true;
    }
    return live;
}

void updateGrid(bool* grid, bool* gridcpy, int w, int h, int A, int B, int C){
    for (int i = 0; i < h; i++) {
        for (int j = 0; j < w; j++) {
            bool judgement = liveOrDie(grid, w, h, i, j, A , B, C);
            gridcpy[i*w + j] = judgement;
        }
    }
}
    
    

    
int main() {
    // reading in grid ------------------//
    int w, h, n, m, A, B, C;
    string line;
    
    cin >> w >> h >> n >> m >> A >> B >> C;
    bool* grid = new bool[w*h];
    bool * gridcpy = new bool[w*h];

    // bool* grid = (bool *) malloc(w*h * sizeof(bool));
    // bool* gridcpy = (bool *) malloc(w*h * sizeof(bool));  
    
    cin.ignore();
    
    for(int i = 0; i < h; ++i){
        getline(cin, line);
        for (int j = 0; j < w; ++j) {
            char c = line[j];
            if(c == '#'){
                grid[i*w + j] = true;
            }
        }
    }
    
    // Updating Grid---------------------//
    for (int i = 0; i < n; i++) {
        
        updateGrid(grid, gridcpy, w, h, A, B, C);
        bool* temp = gridcpy;
        gridcpy = grid;
        grid = temp;
        if (!i%m) printGrid(grid, w, h);
    }

    if (n%m) printGrid(grid, w, h);

    // printGrid(grid, w, h);
    // cout << endl;
    // for (int i = 0; i < 3; i++) {
    //     updateGrid(grid, gridcpy, w, h, A, B, C);
    // } 
    //----------------------------------//

    // free memory 
    delete(grid);
    delete(gridcpy);

    return 0;
}
